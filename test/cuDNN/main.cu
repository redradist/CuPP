#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <memory.hpp>
#include <thread.cuh>
#include <hipDNN.h>
#include <vector>
#include <iostream>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a*x[i] + y[i];
  }

  cuda::barrier();
}

class SaxpyKernel {
 public:
  SaxpyKernel() {
  }

  void compute(int N, float *d_x, float *d_y) {
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  }
};

void convolutionRun() {
  hipdnnHandle_t handle_;
  // Create a cuDNN handle:
  (hipdnnCreate(&handle_));

  // Create your tensor descriptors:
  hipdnnTensorDescriptor_t cudnnIdesc;
  hipdnnFilterDescriptor_t cudnnFdesc;
  hipdnnTensorDescriptor_t cudnnOdesc;
  hipdnnConvolutionDescriptor_t cudnnConvDesc;
  ( hipdnnCreateTensorDescriptor( &cudnnIdesc ));
  ( hipdnnCreateFilterDescriptor( &cudnnFdesc ));
  ( hipdnnCreateTensorDescriptor( &cudnnOdesc ));
  ( hipdnnCreateConvolutionDescriptor( &cudnnConvDesc ));

  // Set NCHW tensor dimensions, not necessarily as multiples of eight (only the input tensor is shown here):
  int dimA[] = {1, 7, 32, 32};
  int strideA[] = {7168, 1024, 32, 1};

  size_t convDim = 2;
  int padA[] = { 1, 1 };
  int convstrideA[] = { 1, 1 };
  int dilationA[] = { 1, 1 };

  ( hipdnnSetTensorNdDescriptor(cudnnIdesc, HIPDNN_DATA_FLOAT,
                                            convDim+2, dimA, strideA) );

  // Allocate and initialize tensors (again, only the input tensor is shown):
  float *alpha;
  float *devPtrI;
  float *devPtrF;
  float *devPtrO;
  float *beta;
  size_t insize = 10000;
  ( hipMalloc((void**)&(alpha), (insize) * sizeof(alpha[0]) ));
  ( hipMalloc((void**)&(devPtrI), (insize) * sizeof(devPtrI[0]) ));
  ( hipMalloc((void**)&(devPtrF), (insize) * sizeof(devPtrF[0]) ));
  ( hipMalloc((void**)&(beta), (insize) * sizeof(beta[0]) ));
  ( hipMalloc((void**)&(devPtrO), (insize) * sizeof(devPtrO[0]) ));

  // No host memory prepared !

  // Set the compute data type (below as HIPDNN_DATA_FLOAT):
  ( hipdnnSetConvolutionNdDescriptor(cudnnConvDesc, convDim, padA, convstrideA, dilationA, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT) );

  // Set the math type to allow cuDNN to use Tensor Cores:
  ( hipdnnSetConvolutionMathType(cudnnConvDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION) );

  // Choose a supported algorithm:
  float *workSpace;
  size_t workSpaceSize;
  hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

  // Allocate your workspace:
  ( hipdnnGetConvolutionForwardWorkspaceSize(handle_, cudnnIdesc,
                                                         cudnnFdesc, cudnnConvDesc,
                                                         cudnnOdesc, algo, &workSpaceSize) );

  if (workSpaceSize > 0) {
    hipMalloc(&workSpace, workSpaceSize);
  }

  // Invoke the convolution:
  auto err = ( hipdnnConvolutionForward(handle_, (void*)(&alpha), cudnnIdesc, devPtrI,
                                         cudnnFdesc, devPtrF, cudnnConvDesc, algo,
                                         workSpace, workSpaceSize, (void*)(&beta),
                                         cudnnOdesc, devPtrO) );
}

int main() {
  hipGraph_t graph;
  hipGraphExec_t instance;
  hipStream_t stream;
  hipError_t err = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  convolutionRun();

  int N = 1 << 20;
  auto x = std::make_unique<float[]>(N);
  auto y = std::make_unique<float[]>(N);

  auto d_x = cuda::makeUnique<float[]>(N);
  auto d_y = cuda::makeUnique<float[]>(N);

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  cuda::memcpy(d_x, x, N*sizeof(float));
  cuda::memcpy(d_y, y, N*sizeof(float));

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  SaxpyKernel().compute(N, d_x.get(), d_y.get());

  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0);
  hipGraphLaunch(instance, stream);
  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);
}
